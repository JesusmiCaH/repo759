#include <iostream>
#include <hip/hip_runtime.h>
#include <random>

__global__ void sum_kernel(int a, int* dA){
	int x = threadIdx.x;
	int y = blockIdx.x;
	dA[y*blockDim.x + x] = a*x + y;
}

int main(){
	unsigned int seed = 759;
	std::mt19937 generator(seed);
	std::uniform_int_distribution<int> dist(0,10);
	int a = dist(generator);
	int blocknum = 2;
	int threadnum = 8;

	int* dA;
        int hA[16];
	hipMalloc(&dA, blocknum*threadnum*sizeof(int));

	sum_kernel<<<blocknum, threadnum>>>(a, dA);
	hipDeviceSynchronize();

	hipMemcpy(hA, dA, blocknum*threadnum*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0; i<blocknum*threadnum; i++){
		printf("%d ",hA[i]);
	}
	std::cout<<std::endl;
	return 0;
}
