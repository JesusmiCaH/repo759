#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <random>
#include "vscale.cuh"

using namespace std;
using std::chrono::duration;

int main(int argc, char* argv[]){
	int n = std::stoi(argv[1]);

	unsigned int seed = 759;
	std::mt19937 generator(seed);
	std::uniform_real_distribution<float> dist_a(-10.0,10.0);
	std::uniform_real_distribution<float> dist_b(0.0,1.0);

	float a[n], b[n];
	float *da, *db;

	hipMalloc(&da, n*sizeof(float));
	hipMalloc(&db, n*sizeof(float));

	// initialize
	for(int i=0; i<n; i++){
		a[i] = dist_a(generator);
		b[i] = dist_b(generator);
	}

	hipMemcpy(da, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, n*sizeof(float), hipMemcpyHostToDevice);
	auto start_time = chrono::steady_clock::now();
	vscale<<<1, n>>>(da, db, n);
	hipDeviceSynchronize();
	auto end_time = chrono::steady_clock::now();

	hipMemcpy(a, da, n*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b, db, n*sizeof(float), hipMemcpyDeviceToHost);

	auto duration_sec = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);	
	cout << duration_sec.count() << endl;
	cout << a[0] << endl;
	cout << a[n-1] << endl;
	
	return 0;
}
