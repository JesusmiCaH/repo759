#include <iostream>
#include <hip/hip_runtime.h>

__global__ void testfunc(){
	int a = threadIdx.x;
	int b = 1;
	for(int i=0; i<=a; i++){
		b*=(i+1);
	}
	printf("%d!=%d\n", a+1, b);
}

int main(){
	int blocknum = 1;
	int threadnum = 8;

	testfunc<<<blocknum, threadnum>>>();
	hipDeviceSynchronize();
	return 0;
}
