#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "vscale.cuh"

__global__ void vscale(float* a, float* b, unsigned int n){
	int id = threadIdx.x;
	if(id < n){
		b[id] = a[id] * b[id];
	}
}

