#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "vscale.cuh"

__global__ void vscale(float* a, float* b, unsigned int n){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id<n)
		b[id] = a[id] * b[id];
}

