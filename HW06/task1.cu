#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <chrono>
#include <random>

using namespace std;
using std::chrono::duration;

int main(int argc, char* argv[]){
    int n = std::stoi(argv[1]);
    unsigned int threads_per_block = std::stoi(argv[2]);

    float *h_A = new float[n * n];
    float *h_B = new float[n * n];
    float *h_C = new float[n * n];

    unsigned int seed = 759;
    std::mt19937 generator(seed);
    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    
    // Initialize
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h_A[i * n + j] = dist(generator);
            h_B[i * n + j] = dist(generator);
        }
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));

    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);
    auto start_time = std::chrono::steady_clock::now();
    matmul(d_A, d_B, d_C, n, threads_per_block);
    auto end_time = std::chrono::steady_clock::now();
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    auto duration_sec = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);	
    
    cout << h_C[n*n-1] << endl;
    cout << duration_sec.count() << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
