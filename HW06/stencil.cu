#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){
    extern __shared__ float shared[];
    float* shared_image = shared;
    float* shared_mask = shared + blockDim.x;
    float* shared_output = shared_mask + (2 * R + 1);

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + tid;

    // Load image and mask
    if(gid < n) {
        // Load shared_image value between idx = R:num_t+R
        shared_image[tid+R] = image[gid];
        if(tid < R){
            // Load shared_image value between idx = 0:R
            if(gid - R >= 0) {
                shared_image[tid] = image[gid - R];
            } else {
                shared_image[tid] = 0.0f; // Handle boundary condition
            }
            // Load shared_image value between idx = R+num_t:2R+num_t
            if(gid + R < n) {
                shared_image[tid + R + blockDim.x] = image[gid + R];
            } else {
                shared_image[tid + R + blockDim.x] = 0.0f; // Handle boundary condition
            }
        }
    }
    if (tid < 2 * R + 1) {
        shared_mask[tid] = mask[tid];
    }

    __syncthreads();

    // Convolution
    if(gid < n){
        shared_output[tid] = 0.0f;
        for(int i = -R; i <= R; i++) {
            shared_output[tid] += shared_image[tid + i + R] * shared_mask[i + R];
        }
        output[gid] = shared_output[tid];
    }
}


__host__ void stencil(const float* image,
    const float* mask,
    float* output,
    unsigned int n,
    unsigned int R,
    unsigned int threads_per_block){
        stencil_kernel<<<(n + threads_per_block - 1) / threads_per_block, threads_per_block, (2 * (R+threads_per_block) + 1) * sizeof(float)>>>(
            image, mask, output, n, R
        );
    }