#include <iostream>
#include <hip/hip_runtime.h>
#include "stencil.cuh"
#include <chrono>

int main(int argc, char* argv[]){
    int n = std::stoi(argv[1]);
    int R = std::stoi(argv[2]);
    unsigned int threads_per_block = 256;

    float* image = new float[n];
    float* mask = new float[2 * R + 1];
    float* output = new float[n];
    unsigned int seed = 759;
    std::mt19937 generator(seed);
    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for (int i = 0; i < n; i++) {
        image[i] = dist(generator);
    }
    for (int i = 0; i < 2 * R + 1; i++) {
        mask[i] = dist(generator);
    }

    float* d_image, *d_mask, *d_output;
    hipMalloc(&d_image, n * sizeof(float));
    hipMalloc(&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    auto start_time = std::chrono::steady_clock::now();
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    auto end_time = std::chrono::steady_clock::now();
    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    auto duration_sec = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);	
    
    std::cout << output[n-1] << std::endl;
    std::cout << duration_sec << std::endl;

    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    delete[] image;
    delete[] mask;
    delete[] output;

    return 0;
}