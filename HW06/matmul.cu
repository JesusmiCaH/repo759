#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n)
{
    // Calculate the row and column index for the element to compute
    int element_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = element_idx / n;
    int col = element_idx % n;

    // Check if the indices are within bounds
    if (row < n && col < n) {
        float value = 0.0f;
        for (int k = 0; k < n; k++) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){
    // Calculate the number of blocks needed
    int num_elements = n * n;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    matmul_kernel<<<num_blocks, threads_per_block>>>(A, B, C, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();
}
