#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <chrono>
#include <random>

using std::chrono::duration;
using namespace std;

template <typename T>
void randomize(T* A, int n, T min, T max) {
    std::random_device rd;
    std::mt19937 gen(rd());
    using DistType = std::conditional_t<
        std::is_integral<T>::value,
        std::uniform_int_distribution<T>,
        std::uniform_real_distribution<T>
        >;
    DistType dist(min, max);
    for (int i = 0; i < n; i++) {
        A[i] = dist(gen);
    }
}

int main(int argc, char* argv[]){
    uint N = std::stoi(argv[1]);
    uint block_dim = std::stoi(argv[2]);

    float * arr = new float[N];
    randomize(arr, N, -1.0f, 1.0f);

    float * d_arr;
    hipMalloc((void**)&d_arr, N * sizeof(float));
    hipMemcpy(d_arr, arr, N * sizeof(float), hipMemcpyHostToDevice);
    float * d_out;
    hipMalloc((void**)&d_out, (N+block_dim-1)/block_dim * sizeof(float));

    auto start_time = std::chrono::steady_clock::now();
    reduce(&d_arr, &d_out, N, block_dim);
    auto end_time = std::chrono::steady_clock::now();
    auto duration_sec = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);

    hipMemcpy(arr, d_arr, sizeof(float), hipMemcpyDeviceToHost);
    cout << arr[0] << endl;
    cout << duration_sec.count() << endl;
    cout << "------------------------" << endl;

    return 0;
}
