#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "matmul.cuh"

template <typename T, unsigned int block_dim>
__global__ void matmul_kernel(
    const T *A, T *B, T *C, 
    unsigned int n
) {
    __shared__ T tile_A[block_dim][block_dim];
    __shared__ T tile_B[block_dim][block_dim];
    uint round = (n + block_dim - 1) / block_dim;
    uint row = blockIdx.y * block_dim + threadIdx.y;
    uint col = blockIdx.x * block_dim + threadIdx.x;
    C[row * n + col] = 0;
    for(int i = 0; i < round; i++){
        if (i*block_dim + threadIdx.x < n && row < n) {
            tile_A[threadIdx.y][threadIdx.x] = A[row * n + i * block_dim + threadIdx.x];
        } else {
            tile_A[threadIdx.y][threadIdx.x] = 0;
        }
        if (i*block_dim + threadIdx.y < n && col < n) {
            tile_B[threadIdx.y][threadIdx.x] = B[(i * block_dim + threadIdx.y) * n + col];
        } else {
            tile_B[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (int j = 0; j<block_dim; j++){
            C[row * n + col] += tile_A[threadIdx.y][j] * tile_B[j][threadIdx.x];
        }
        __syncthreads();
    }
}

__host__ void matmul_1(
    const int *A, const int *B, int *C, 
    unsigned int n, unsigned int block_dim
) {
    int *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul_kernel<int, block_dim> <<<grid, block>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__host__ void matmul_2(
    const float *A, const float *B, float *C, 
    unsigned int n, unsigned int block_dim
){
    float *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(float);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul_kernel<float, block_dim> <<<grid, block>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__host__ void matmul_3(
    const double *A, const double *B, double *C, 
    unsigned int n, unsigned int block_dim
){
    double *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(double);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul_kernel<double, block_dim> <<<grid, block>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
    