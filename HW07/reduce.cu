#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "reduce.cuh"

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    if (i < n) {
        sdata[tid] = g_idata[i];
        if (i+blockDim.x < n) {
            sdata[tid] += g_idata[i + blockDim.x];
        }
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

__host__ void reduce(float **input, float **output, unsigned int N,
    unsigned int threads_per_block
){
    unsigned int blocks = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);
    reduce_kernel<<<blocks, threads_per_block, threads_per_block * sizeof(float)>>>(input[0], output[0], N);
    hipDeviceSynchronize();
    if (blocks > 1) {
        reduce(output, input, blocks, threads_per_block);
    }
    if (blocks == 1) {
        hipMemcpy(input[0], output[0], sizeof(float), hipMemcpyDeviceToDevice);
    }
}
