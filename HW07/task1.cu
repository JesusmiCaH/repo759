#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <chrono>
#include <random>

using std::chrono::duration;
using namespace std;

template <typename T>
void randomize(T* A, int n, T min, T max) {
    std::random_device rd;
    std::mt19937 gen(rd());
    using DistType = std::conditional_t<
   	std::is_integral<T>::value,
    	std::uniform_int_distribution<T>,
    	std::uniform_real_distribution<T>
	>;
    DistType dist(min, max);
    for (int i = 0; i < n; i++) {
        A[i] = dist(gen);
    }
}

int main(int argc, char* argv[]){
    uint n = std::stoi(argv[1]);
    uint block_dim = std::stoi(argv[2]);

    int* A_1 = new int[n * n];
    int* B_1 = new int[n * n];
    int* C_1 = new int[n * n];
    float* A_2 = new float[n * n];
    float* B_2 = new float[n * n];
    float* C_2 = new float[n * n];
    double* A_3 = new double[n * n];
    double* B_3 = new double[n * n];
    double* C_3 = new double[n * n];
    randomize(A_1, n*n, -10, 10);
    randomize(B_1, n*n, -10, 10);
    randomize(A_2, n*n, -10.0f, 10.0f);
    randomize(B_2, n*n, -10.0f, 10.0f);
    randomize(A_3, n*n, -10.0, 10.0);
    randomize(B_3, n*n, -10.0, 10.0);
    

    auto start_time = std::chrono::steady_clock::now();
    matmul_1(A_1, B_1, C_1, n, block_dim);
    auto end_time = std::chrono::steady_clock::now();
    auto duration_sec_1 = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);	

    start_time = std::chrono::steady_clock::now();
    matmul_2(A_2, B_2, C_2, n, block_dim);
    end_time = std::chrono::steady_clock::now();
    auto duration_sec_2 = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);

    start_time = std::chrono::steady_clock::now();
    matmul_3(A_3, B_3, C_3, n, block_dim);
    end_time = std::chrono::steady_clock::now();
    auto duration_sec_3 = chrono::duration_cast<duration<double, std::milli>>(end_time - start_time);
    
    cout << C_1[0] << endl << C_1[n-1] << endl;
    cout << duration_sec_1.count() << endl;
    cout << C_2[0] << endl << C_2[n-1] << endl;
    cout << duration_sec_2.count() << endl;
    cout << C_3[0] << endl << C_3[n-1] << endl;
    cout << duration_sec_3.count() << endl;
    cout << "------------------------" << endl;

    return 0;
}
